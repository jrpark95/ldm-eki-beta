#include "hip/hip_runtime.h"
/******************************************************************************
 * @file ldm_init_config.cu
 * @brief Configuration loading and initialization for LDM simulation system
 *
 * @details Implements configuration parsers for all input files:
 *          - setting.txt: Core simulation parameters
 *          - source.txt: Emission source locations and release cases
 *          - eki_settings.txt: Ensemble Kalman Inversion parameters
 *          - Modernized config files: simulation.conf, physics.conf, etc.
 *
 * @note Legacy file support maintained for backward compatibility
 * @note New modular config system introduced in 2025-10-17 (Phase 1)
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/

#include "../core/ldm.cuh"
#include "../physics/ldm_nuclides.cuh"
#include "colors.h"

/******************************************************************************
 * @brief Load simulation configuration from legacy setting.txt file
 *
 * @details Parses input/setting.txt to load core simulation parameters and
 *          source.txt for emission source definitions. This is the legacy
 *          configuration loader maintained for backward compatibility.
 *
 *          Configuration includes:
 *          - Temporal parameters: time_end, dt, output frequency
 *          - Particle properties: count, size distribution, density
 *          - Physics model switches: turbulence, deposition, decay
 *          - Atmospheric conditions: rural/urban, stability parameterization
 *          - Meteorological data source: GFS/LDAPS selection
 *          - Source locations: coordinates (lon, lat, height)
 *          - Release cases: emission values per source/time
 *
 * @pre Input files must exist:
 *      - input/setting.txt (simulation parameters)
 *      - input/source.txt (source locations and release cases)
 *      - cram/A60.csv (CRAM decay matrix if radioactive decay enabled)
 *
 * @post Member variables populated:
 *       - time_end, dt, freq_output, nop
 *       - isRural, isPG, isGFS
 *       - sources vector, concentrations vector
 *       - decayConstants, drydepositionVelocity vectors
 * @post Physics model switches set: g_turb_switch, g_drydep, g_wetdep, g_raddecay
 * @post CRAM system initialized (if radioactive decay enabled)
 * @post Output directory cleaned
 *
 * @algorithm
 *   1. Load setting.txt using ConfigReader
 *   2. Parse simulation parameters (time, particle count, etc.)
 *   3. Load physics model switches
 *   4. Parse species properties (decay constants, sizes, densities)
 *   5. Initialize CRAM decay system with dt from config
 *   6. Open source.txt for emission source parsing
 *   7. Parse [SOURCE] section: lon, lat, height coordinates
 *   8. Parse [SOURCE_TERM] section: decay constants, deposition velocities
 *   9. Parse [RELEASE_CASES] section: location, source term, emission value
 *  10. Close file and clean output directory
 *
 * @note Configuration values passed to GPU kernels via KernelScalars struct
 * @note No longer uses __constant__ memory (removed in refactoring)
 *
 * @see loadSimulationConfig() for modernized config file loader
 * @see input/setting.txt for legacy file format specification
 * @see input/source.txt for source definition format
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::loadSimulationConfiguration(){

    if (!g_config.loadConfig("input/setting.txt")) {
        std::cerr << "Failed to load configuration file" << std::endl;
        exit(1);
    }

    FILE* sourceFile;

    // Parse temporal parameters
    time_end = g_config.getFloat("Time_end(s)", 64800.0f);      // Simulation duration (seconds)
    dt = g_config.getFloat("dt(s)", 10.0f);                     // Time step (seconds)
    freq_output = g_config.getInt("Plot_output_freq", 10);      // VTK output frequency

    // Parse particle parameters
    nop = g_config.getInt("Total_number_of_particle", 10000);   // Total particle count

    // Hardcode atmospheric conditions and meteorological data (v1.0 production settings)
    isRural = 1;  // HARDCODED: Rural conditions
    isPG = 1;     // HARDCODED: Pasquill-Gifford stability scheme
    isGFS = 1;    // HARDCODED: GFS meteorological data

    // Load terminal output settings
    g_sim.fixedScrollOutput = g_config.getInt("fixed_scroll_output", 1);

    // Hardcode turbulence model (not implemented in v1.0)
    g_turb_switch = 0;  // HARDCODED: Turbulence model not implemented

    // Note: Other physics models (dry/wet deposition, radioactive decay)
    // are loaded from physics.conf in loadPhysicsConfig()

    // Clean output directory before simulation
    cleanOutputDirectory();

    // Parse species properties (up to 4 species supported)
    std::vector<std::string> species_names = g_config.getStringArray("species_names");
    std::vector<float> decay_constants = g_config.getFloatArray("decay_constants");
    std::vector<float> deposition_velocities = g_config.getFloatArray("deposition_velocities");
    std::vector<float> particle_sizes = g_config.getFloatArray("particle_sizes");
    std::vector<float> particle_densities = g_config.getFloatArray("particle_densities");
    std::vector<float> size_standard_deviations = g_config.getFloatArray("size_standard_deviations");

    for (int i = 0; i < 4 && i < species_names.size(); i++) {
        g_mpi.species[i] = species_names[i];
        g_mpi.decayConstants[i] = (i < decay_constants.size()) ? decay_constants[i] : 1.00e-6f;
        g_mpi.depositionVelocities[i] = (i < deposition_velocities.size()) ? deposition_velocities[i] : 0.01f;
        g_mpi.particleSizes[i] = (i < particle_sizes.size()) ? particle_sizes[i] : 0.6f;
        g_mpi.particleDensities[i] = (i < particle_densities.size()) ? particle_densities[i] : 2500.0f;
        g_mpi.sizeStandardDeviations[i] = (i < size_standard_deviations.size()) ? size_standard_deviations[i] : 0.01f;
    }

    // Initialize CRAM decay system with dynamic dt from configuration
    if (initialize_cram_system("cram/A60.csv")) {
        // Successfully computed exp(-A*dt) matrix for CRAM decay
    } else {
        std::cerr << "Warning: CRAM system initialization failed, using traditional decay" << std::endl;
    }

    // Open source configuration file
    std::string source_file_path = g_config.getString("input_base_path", "./input/") + "source.txt";
    sourceFile = fopen(source_file_path.c_str(), "r");

    if (!sourceFile){
        std::cerr << "Failed to open source.txt" << std::endl;
        exit(1);
    }

    // Parse source.txt with three sections: [SOURCE], [SOURCE_TERM], [RELEASE_CASES]
    char buffer[256];
    while (fgets(buffer, sizeof(buffer), sourceFile)){
        if (buffer[0] == '#') continue;  // Skip comments

        // Parse [SOURCE] section: source coordinates (lon, lat, height)
        if (strstr(buffer, "[SOURCE]")) {
            while (fgets(buffer, sizeof(buffer), sourceFile) && !strstr(buffer, "[SOURCE_TERM]")) {
                if (buffer[0] == '#') continue;

                Source src;
                sscanf(buffer, "%f %f %f", &src.lon, &src.lat, &src.height);
                sources.push_back(src);
            }
            sources.pop_back();  // Remove sentinel entry
        }

        // Parse [SOURCE_TERM] section: decay constants and deposition velocities
        if (strstr(buffer, "[SOURCE_TERM]")){
            while (fgets(buffer, sizeof(buffer), sourceFile) && !strstr(buffer, "[RELEASE_CASES]")) {
                if (buffer[0] == '#') continue;

                int srcnum;
                float decay, depvel;
                sscanf(buffer, "%d %f %f", &srcnum, &decay, &depvel);
                decayConstants.push_back(decay);
                drydepositionVelocity.push_back(depvel);
            }
            decayConstants.pop_back();      // Remove sentinel entry
            drydepositionVelocity.pop_back();
        }

        // Parse [RELEASE_CASES] section: emission scenarios
        if (strstr(buffer, "[RELEASE_CASES]")){
            while (fgets(buffer, sizeof(buffer), sourceFile)) {
                if (buffer[0] == '#') continue;
                Concentration conc;
                sscanf(buffer, "%d %d %lf", &conc.location, &conc.sourceterm, &conc.value);
                concentrations.push_back(conc);
            }
        }
    }

    fclose(sourceFile);

    // Note: Configuration values now passed via KernelScalars struct to kernels
    // __constant__ memory symbols removed during 2025 refactoring (non-RDC compatibility)

}
/******************************************************************************
 * @brief Clean output directory before simulation starts
 *
 * @details Removes previous run artifacts from output/ directory to prevent
 *          data contamination between simulation runs. Platform-specific
 *          implementation using system calls.
 *
 * @post output/ directory cleared of:
 *       - *.vtk files (VTK particle visualization)
 *       - *.csv files (validation data)
 *       - *.txt files (text output)
 *
 * @note Platform-specific behavior:
 *       - Windows: Uses 'del /Q output\*.*' command
 *       - Linux/macOS: Uses 'rm -f output/*.{vtk,csv,txt}' commands
 * @note Errors suppressed (2>nul on Windows, 2>/dev/null on Unix)
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::cleanOutputDirectory() {
    std::cout << "Cleaning output directory... " << std::flush;

    // Remove all output files using platform-specific commands
    #ifdef _WIN32
        system("del /Q output\\*.* 2>nul");
    #else
        system("rm -f output/*.vtk 2>/dev/null");
        system("rm -f output/*.csv 2>/dev/null");
        system("rm -f output/*.txt 2>/dev/null");
    #endif

    std::cout << Color::GREEN << "" << Color::RESET << std::endl;
}

/******************************************************************************
 * @brief Load Ensemble Kalman Inversion settings from eki_settings.txt
 *
 * @details Parses input/eki_settings.txt to configure the EKI optimization
 *          framework. Uses a state machine to parse multi-line sections
 *          (receptor locations, emission time series) and key-value pairs.
 *
 *          Configuration includes:
 *          - Receptor definitions: locations (lat/lon), capture radius
 *          - Emission time series: true values (for observations), prior guess
 *          - EKI algorithm parameters: ensemble size, iterations, noise level
 *          - Algorithm variants: adaptive step size, localization, regularization
 *          - GPU acceleration settings: forward/inverse model GPU usage
 *          - Debug options: Memory Doctor mode for IPC diagnostics
 *
 * @pre input/eki_settings.txt must exist
 * @pre EKI mode must be enabled (function called when g_eki.mode = true)
 *
 * @post g_eki struct fully populated with EKI parameters
 * @post g_eki.receptor_locations: vector of (lat, lon) pairs
 * @post g_eki.true_emissions: time series for generating observations
 * @post g_eki.prior_emissions: initial guess for optimization
 * @post Algorithm switches set: adaptive_eki, localized_eki, regularization
 *
 * @algorithm State machine parser:
 *   1. Initialize g_eki with default values
 *   2. Parse file line by line:
 *      - Section headers toggle state flags:
 *        * RECEPTOR_LOCATIONS_MATRIX= → read receptor coordinates
 *        * TRUE_EMISSION_SERIES= → read true emission values
 *        * PRIOR_EMISSION_SERIES= → read prior emission values
 *      - Key-value pairs (KEY=VALUE) reset state flags and parse parameters
 *      - Matrix data lines parsed according to current state
 *   3. Validate configuration (e.g., num_receptors matches location count)
 *   4. Print essential EKI configuration summary
 *
 * @note File format:
 *       - Comments: Lines starting with #
 *       - Key-value: KEY=VALUE (no spaces around =)
 *       - Matrix sections: Header line followed by data lines
 * @note State machine ensures correct parsing of multi-line sections
 *
 * @see input/eki_settings.txt for configuration file format
 * @see src/eki/RunEstimator.py for Python EKI executor
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::loadReceptorConfig() {
    // Try modern config file first, fallback to legacy (embedded in eki_settings.txt)
    FILE* receptorFile = fopen("input/receptor.conf", "r");
    const char* config_filename = "input/receptor.conf";

    if (!receptorFile) {
        receptorFile = fopen("input/eki_settings.txt", "r");
        config_filename = "input/eki_settings.txt";
    }

    if (!receptorFile) {
        std::cerr << "\n" << Color::RED << Color::BOLD << "[ERROR] " << Color::RESET
                  << "Failed to open receptor configuration file" << std::endl;
        std::cerr << "  Tried: input/receptor.conf, input/eki_settings.txt" << std::endl;
        exit(1);
    }

    std::cout << "\n" << Color::CYAN << "[SYSTEM] " << Color::RESET
              << "Loading receptor settings from " << Color::BOLD << config_filename << Color::RESET << "... " << std::flush;

    char buffer[256];

    // Initialize receptor configuration
    g_eki.num_receptors = 0;
    g_eki.receptor_locations.clear();

    // State machine flag for multi-line section parsing
    bool reading_receptor_locations = false;

    while (fgets(buffer, sizeof(buffer), receptorFile)) {
        // Skip comments and empty lines
        if (buffer[0] == '#' || buffer[0] == '\n' || buffer[0] == '\r') {
            continue;
        }

        // Normalize separator: convert ':' to '=' for uniform parsing
        char* colon_pos = strchr(buffer, ':');
        if (colon_pos && !strchr(buffer, '=')) {
            *colon_pos = '=';
        }

        // Check for multi-line section header
        if (strstr(buffer, "RECEPTOR_LOCATIONS=") || strstr(buffer, "RECEPTOR_LOCATIONS_MATRIX=")) {
            reading_receptor_locations = true;
            continue;
        }

        // Reset section flag when encountering key-value pairs
        if (strchr(buffer, '=') != nullptr) {
            reading_receptor_locations = false;
        }

        // Parse receptor location data
        if (reading_receptor_locations) {
            float lat, lon;
            if (sscanf(buffer, "%f %f", &lat, &lon) == 2) {
                g_eki.receptor_locations.push_back(std::make_pair(lat, lon));
            }
        }

        // Parse key-value pairs
        if (strchr(buffer, '=') != nullptr) {
            if (strstr(buffer, "NUM_RECEPTORS=")) {
                sscanf(buffer, "NUM_RECEPTORS=%d", &g_eki.num_receptors);
            }
            else if (strstr(buffer, "RECEPTOR_CAPTURE_RADIUS=")) {
                sscanf(buffer, "RECEPTOR_CAPTURE_RADIUS=%f", &g_eki.receptor_capture_radius);
            }
        }
    }

    fclose(receptorFile);

    // ========== COMPREHENSIVE VALIDATION ==========

    // ===== VALIDATION: NUM_RECEPTORS =====
    if (g_eki.num_receptors <= 0) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid NUM_RECEPTORS: " << g_eki.num_receptors << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    At least one receptor must be defined for EKI mode." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    NUM_RECEPTORS >= 1" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " 3-10 receptors for good spatial coverage" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    if (g_eki.receptor_locations.size() != static_cast<size_t>(g_eki.num_receptors)) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Receptor count mismatch" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    NUM_RECEPTORS=" << g_eki.num_receptors
                  << " but " << g_eki.receptor_locations.size()
                  << " receptor locations defined" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Solution:" << Color::RESET << std::endl;
        std::cerr << "    Ensure RECEPTOR_LOCATIONS has exactly "
                  << g_eki.num_receptors << " lines" << std::endl;
        std::cerr << "    Format: latitude longitude (one per line)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: Receptor locations =====
    for (size_t i = 0; i < g_eki.receptor_locations.size(); i++) {
        float lat = g_eki.receptor_locations[i].first;
        float lon = g_eki.receptor_locations[i].second;

        if (lat < -90.0f || lat > 90.0f) {
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Invalid receptor latitude: " << lat << "° (receptor "
                      << (i+1) << ")" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Required range:" << Color::RESET << std::endl;
            std::cerr << "    -90.0 <= latitude <= 90.0 (degrees)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                      << " " << config_filename << ", RECEPTOR_LOCATIONS" << std::endl;
            std::cerr << std::endl;
            exit(1);
        }

        if (lon < -180.0f || lon > 180.0f) {
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Invalid receptor longitude: " << lon << "° (receptor "
                      << (i+1) << ")" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Required range:" << Color::RESET << std::endl;
            std::cerr << "    -180.0 <= longitude <= 180.0 (degrees)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                      << " " << config_filename << ", RECEPTOR_LOCATIONS" << std::endl;
            std::cerr << std::endl;
            exit(1);
        }
    }

    // ===== VALIDATION: RECEPTOR_CAPTURE_RADIUS =====
    if (g_eki.receptor_capture_radius <= 0.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid RECEPTOR_CAPTURE_RADIUS: "
                  << g_eki.receptor_capture_radius << "°" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Capture radius must be positive." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    RECEPTOR_CAPTURE_RADIUS > 0.0 (degrees)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Typical values:" << Color::RESET << std::endl;
        std::cerr << "    - Fine resolution:   0.01° (~1 km)" << std::endl;
        std::cerr << "    - Standard:          0.025° (~2.5 km)" << std::endl;
        std::cerr << "    - Coarse:            0.05° (~5 km)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (g_eki.receptor_capture_radius > 1.0f) {
        std::cerr << std::endl << Color::YELLOW << Color::BOLD << "[WARNING] "
                  << Color::RESET << "Very large RECEPTOR_CAPTURE_RADIUS: "
                  << g_eki.receptor_capture_radius << "° (~"
                  << (g_eki.receptor_capture_radius * 111.0f) << " km)" << std::endl;
        std::cerr << "  This may capture particles from large areas, reducing spatial resolution." << std::endl;
        std::cerr << "  Consider using smaller radius (0.01-0.05°) for better accuracy." << std::endl;
        std::cerr << std::endl;
    }

    std::cout << "done\n";
}

void LDM::loadEKISettings() {
    // Try modern config file first, fallback to legacy
    FILE* ekiFile = fopen("input/eki.conf", "r");
    const char* config_filename = "input/eki.conf";

    if (!ekiFile) {
        ekiFile = fopen("input/eki_settings.txt", "r");
        config_filename = "input/eki_settings.txt";
    }

    if (!ekiFile) {
        std::cerr << "\n" << Color::RED << Color::BOLD << "[ERROR] " << Color::RESET
                  << "Failed to open EKI configuration file" << std::endl;
        std::cerr << "  Tried: input/eki.conf, input/eki_settings.txt" << std::endl;
        exit(1);
    }

    std::cout << "\n" << Color::CYAN << "[SYSTEM] " << Color::RESET
              << "Loading EKI settings from " << Color::BOLD << config_filename << Color::RESET << "... " << std::flush;

    char buffer[256];

    // Initialize global EKI configuration - NO DEFAULTS, all values must be in config file
    g_eki.mode = true;  // EKI mode is enabled by calling this function

    // Clear emission vectors
    g_eki.true_emissions.clear();
    g_eki.prior_emissions.clear();

    // Initialize to sentinel values to detect missing parameters
    g_eki.time_interval = -1.0f;          // Must be set by config
    g_eki.time_unit = "";                 // Must be set by config
    g_eki.prior_mode = "";                // Must be set by config
    g_eki.prior_constant = -1.0f;         // Must be set by config (if prior_mode = constant)
    g_eki.ensemble_size = -1;             // Must be set by config
    g_eki.noise_level = -1.0f;            // Must be set by config
    g_eki.iteration = -1;                 // Must be set by config
    g_eki.perturb_option = "";            // Must be set by config
    g_eki.adaptive_eki = "";              // Must be set by config
    g_eki.localized_eki = "";             // Must be set by config
    g_eki.regularization = "";            // Must be set by config
    g_eki.renkf_lambda = -1.0f;           // Must be set by config (if regularization = on)

    // State machine flags for multi-line section parsing
    bool reading_true_emissions = false;
    bool reading_prior_emissions = false;
    
    while (fgets(buffer, sizeof(buffer), ekiFile)) {
        // Skip comments and empty lines
        if (buffer[0] == '#' || buffer[0] == '\n' || buffer[0] == '\r') {
            continue;
        }

        // Normalize separator: convert ':' to '=' for uniform parsing
        // This allows both "KEY: value" (new format) and "KEY=value" (legacy format)
        char* colon_pos = strchr(buffer, ':');
        if (colon_pos && !strchr(buffer, '=')) {
            *colon_pos = '=';
        }

        // State machine: Check for multi-line section headers
        if (strstr(buffer, "TRUE_EMISSION_SERIES=")) {
            reading_true_emissions = true;
            reading_prior_emissions = false;
            continue;
        }

        if (strstr(buffer, "PRIOR_EMISSION_SERIES=")) {
            reading_true_emissions = false;
            reading_prior_emissions = true;
            continue;
        }

        // Reset section flags when encountering key-value pairs
        if (strchr(buffer, '=') != nullptr) {
            reading_true_emissions = false;
            reading_prior_emissions = false;
        }

        // Parse matrix data based on current state
        if (reading_true_emissions) {
            float emission;
            if (sscanf(buffer, "%f", &emission) == 1) {
                g_eki.true_emissions.push_back(emission);
            }
        }
        else if (reading_prior_emissions) {
            float emission;
            if (sscanf(buffer, "%f", &emission) == 1) {
                g_eki.prior_emissions.push_back(emission);
            }
        }
        
        // Parse key-value pairs (section flags already reset above)
        if (strchr(buffer, '=') != nullptr) {

            // Temporal parameters
            if (strstr(buffer, "EKI_TIME_INTERVAL=")) {
                sscanf(buffer, "EKI_TIME_INTERVAL=%f", &g_eki.time_interval);
            }
            else if (strstr(buffer, "EKI_TIME_UNIT=")) {
                char unit[32];
                sscanf(buffer, "EKI_TIME_UNIT=%s", unit);
                g_eki.time_unit = std::string(unit);
            }

            // Prior emission settings
            else if (strstr(buffer, "PRIOR_EMISSION_MODE=")) {
                char mode[32];
                sscanf(buffer, "PRIOR_EMISSION_MODE=%s", mode);
                g_eki.prior_mode = std::string(mode);
            }
            else if (strstr(buffer, "PRIOR_EMISSION_CONSTANT=")) {
                sscanf(buffer, "PRIOR_EMISSION_CONSTANT=%f", &g_eki.prior_constant);
            }

            // EKI algorithm parameters
            else if (strstr(buffer, "EKI_ENSEMBLE_SIZE=")) {
                sscanf(buffer, "EKI_ENSEMBLE_SIZE=%d", &g_eki.ensemble_size);
            }
            else if (strstr(buffer, "EKI_NOISE_LEVEL=")) {
                sscanf(buffer, "EKI_NOISE_LEVEL=%f", &g_eki.noise_level);
            }
            else if (strstr(buffer, "EKI_ITERATION=")) {
                sscanf(buffer, "EKI_ITERATION=%d", &g_eki.iteration);
            }
            else if (strstr(buffer, "EKI_PERTURB_OPTION=")) {
                char opt[32];
                sscanf(buffer, "EKI_PERTURB_OPTION=%s", opt);
                g_eki.perturb_option = std::string(opt);
            }

            // EKI algorithm variants
            else if (strstr(buffer, "EKI_ADAPTIVE=")) {
                char opt[32];
                sscanf(buffer, "EKI_ADAPTIVE=%s", opt);
                g_eki.adaptive_eki = std::string(opt);
            }
            else if (strstr(buffer, "EKI_LOCALIZED=")) {
                char opt[32];
                sscanf(buffer, "EKI_LOCALIZED=%s", opt);
                g_eki.localized_eki = std::string(opt);
            }
            else if (strstr(buffer, "EKI_REGULARIZATION=")) {
                char opt[32];
                sscanf(buffer, "EKI_REGULARIZATION=%s", opt);
                g_eki.regularization = std::string(opt);
            }
            else if (strstr(buffer, "EKI_RENKF_LAMBDA=")) {
                sscanf(buffer, "EKI_RENKF_LAMBDA=%f", &g_eki.renkf_lambda);
            }

            // Debug mode
            else if (strstr(buffer, "MEMORY_DOCTOR_MODE=")) {
                char mode[32];
                sscanf(buffer, "MEMORY_DOCTOR_MODE=%s", mode);
                g_eki.memory_doctor_mode = (strcmp(mode, "On") == 0 || strcmp(mode, "on") == 0 ||
                                           strcmp(mode, "ON") == 0 || strcmp(mode, "1") == 0);
            }
        }
    }
    
    fclose(ekiFile);

    // ========== COMPREHENSIVE VALIDATION ==========

    // ===== VALIDATION: EKI_TIME_UNIT (must be set) =====
    if (g_eki.time_unit.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: EKI_TIME_UNIT" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Time unit for emission time series must be specified." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_TIME_UNIT: <string>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    - \"seconds\"" << std::endl;
        std::cerr << "    - \"minutes\"" << std::endl;
        std::cerr << "    - \"hours\"" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    EKI_TIME_UNIT: minutes" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: PRIOR_EMISSION_MODE (must be set) =====
    if (g_eki.prior_mode.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: PRIOR_EMISSION_MODE" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Prior emission mode must be specified for EKI." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    PRIOR_EMISSION_MODE: <string>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    - \"constant\" (single value for all timesteps)" << std::endl;
        std::cerr << "    - \"series\" (use PRIOR_EMISSION_SERIES)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    PRIOR_EMISSION_MODE: constant" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: PRIOR_EMISSION_CONSTANT (if mode = constant) =====
    if (g_eki.prior_mode == "constant" && g_eki.prior_constant < 0.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: PRIOR_EMISSION_CONSTANT" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    PRIOR_EMISSION_MODE is set to 'constant' but" << std::endl;
        std::cerr << "    PRIOR_EMISSION_CONSTANT is not specified." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    PRIOR_EMISSION_CONSTANT: <positive number> (Bq/s)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Typical values:" << Color::RESET << std::endl;
        std::cerr << "    - Small source:  1.0e+8 Bq/s" << std::endl;
        std::cerr << "    - Medium source: 1.0e+10 Bq/s" << std::endl;
        std::cerr << "    - Large source:  1.0e+12 Bq/s" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    PRIOR_EMISSION_CONSTANT: 1.5e+8" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: EKI_PERTURB_OPTION (must be set) =====
    if (g_eki.perturb_option.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: EKI_PERTURB_OPTION" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Perturbation option for ensemble generation must be specified." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_PERTURB_OPTION: <string>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    - \"observations\" (perturb observations - recommended)" << std::endl;
        std::cerr << "    - \"parameters\" (perturb state parameters)" << std::endl;
        std::cerr << "    - \"both\" (perturb both)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " observations" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    EKI_PERTURB_OPTION: observations" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: EKI_ADAPTIVE (must be set) =====
    if (g_eki.adaptive_eki.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: EKI_ADAPTIVE" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Adaptive step size option must be specified." << std::endl;
        std::cerr << "    This controls whether the EKI algorithm uses adaptive step sizing." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_ADAPTIVE: <On|Off>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " On (improves convergence)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    EKI_ADAPTIVE: On" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: EKI_LOCALIZED (must be set) =====
    if (g_eki.localized_eki.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: EKI_LOCALIZED" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Covariance localization option must be specified." << std::endl;
        std::cerr << "    This controls whether to remove spurious correlations in ensemble." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_LOCALIZED: <On|Off>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " On (for ensemble_size < 100)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    EKI_LOCALIZED: On" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: EKI_REGULARIZATION (must be set) =====
    if (g_eki.regularization.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: EKI_REGULARIZATION" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Regularization option must be specified." << std::endl;
        std::cerr << "    This controls whether to use Tikhonov regularization." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_REGULARIZATION: <On|Off>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " Off (unless ill-posed problem)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    EKI_REGULARIZATION: Off" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: EKI_RENKF_LAMBDA (if regularization = On) =====
    if ((g_eki.regularization == "On" || g_eki.regularization == "on") && g_eki.renkf_lambda < 0.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: EKI_RENKF_LAMBDA" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    EKI_REGULARIZATION is set to 'On' but" << std::endl;
        std::cerr << "    EKI_RENKF_LAMBDA (regularization parameter) is not specified." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_RENKF_LAMBDA: <positive number>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Typical values:" << Color::RESET << std::endl;
        std::cerr << "    - Weak regularization:   0.01-0.1" << std::endl;
        std::cerr << "    - Medium regularization: 0.1-1.0" << std::endl;
        std::cerr << "    - Strong regularization: 1.0-10.0" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    EKI_RENKF_LAMBDA: 0.1" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: EKI_ENSEMBLE_SIZE =====
    if (g_eki.ensemble_size <= 0) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid EKI_ENSEMBLE_SIZE: " << g_eki.ensemble_size << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Ensemble size must be positive." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_ENSEMBLE_SIZE >= 10 (minimum for Kalman filter)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended range:" << Color::RESET << std::endl;
        std::cerr << "    - Quick test:  20-50 members" << std::endl;
        std::cerr << "    - Standard:    50-100 members (good balance)" << std::endl;
        std::cerr << "    - High quality: 100-500 members" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (g_eki.ensemble_size < 10) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Too few ensemble members: " << g_eki.ensemble_size << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Ensemble Kalman methods require sufficient members to estimate" << std::endl;
        std::cerr << "    covariance matrices. < 10 members produces unreliable results." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_ENSEMBLE_SIZE >= 10" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " At least 50 members" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (g_eki.ensemble_size > 10000) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Excessive ensemble size: " << g_eki.ensemble_size << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    This will cause:" << std::endl;
        std::cerr << "    - Enormous memory consumption" << std::endl;
        std::cerr << "    - Extremely long computation times" << std::endl;
        std::cerr << "    - Diminishing returns in accuracy" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_ENSEMBLE_SIZE <= 10000" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Practical maximum:" << Color::RESET << " 500 members" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: EKI_ITERATION =====
    if (g_eki.iteration <= 0) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid EKI_ITERATION: " << g_eki.iteration << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    At least one iteration is required." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_ITERATION >= 1" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Typical values:" << Color::RESET << std::endl;
        std::cerr << "    - Quick test:  1-3 iterations" << std::endl;
        std::cerr << "    - Standard:    3-5 iterations" << std::endl;
        std::cerr << "    - Convergence: 5-10 iterations" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (g_eki.iteration > 100) {
        std::cerr << std::endl << Color::YELLOW << Color::BOLD << "[WARNING] "
                  << Color::RESET << "Very many iterations: " << g_eki.iteration << std::endl;
        std::cerr << "  This will require extremely long computation time." << std::endl;
        std::cerr << "  Consider using fewer iterations (e.g., 3-10) with convergence checking." << std::endl;
        std::cerr << std::endl;
    }

    // ===== VALIDATION: EKI_NOISE_LEVEL =====
    if (g_eki.noise_level < 0.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid EKI_NOISE_LEVEL: " << g_eki.noise_level << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Noise level cannot be negative." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_NOISE_LEVEL >= 0.0 (fraction, e.g., 0.1 = 10%)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Typical values:" << Color::RESET << std::endl;
        std::cerr << "    - Low noise:     0.01-0.05 (1-5%)" << std::endl;
        std::cerr << "    - Medium noise:  0.05-0.10 (5-10%)" << std::endl;
        std::cerr << "    - High noise:    0.10-0.20 (10-20%)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (g_eki.noise_level > 1.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Excessive EKI_NOISE_LEVEL: " << g_eki.noise_level
                  << " (" << (g_eki.noise_level * 100.0f) << "%)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Noise level > 100% indicates measurement error exceeds signal." << std::endl;
        std::cerr << "    This makes inverse problem ill-posed." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_NOISE_LEVEL <= 1.0 (100%)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " 0.05-0.15 (5-15%)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: TRUE_EMISSION_SERIES =====
    if (g_eki.true_emissions.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "No TRUE_EMISSION_SERIES data found" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    TRUE_EMISSION_SERIES must have at least one time step." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Solution:" << Color::RESET << std::endl;
        std::cerr << "    Define emission values in TRUE_EMISSION_SERIES section" << std::endl;
        std::cerr << "    One value per line (in Bq)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    TRUE_EMISSION_SERIES=" << std::endl;
        std::cerr << "    1.0e+12" << std::endl;
        std::cerr << "    1.0e+12" << std::endl;
        std::cerr << "    5.0e+11" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // Check for negative or unrealistic emission values
    for (size_t i = 0; i < g_eki.true_emissions.size(); i++) {
        if (g_eki.true_emissions[i] < 0.0f) {
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Negative emission value at timestep " << (i+1)
                      << ": " << g_eki.true_emissions[i] << " Bq" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Emission rates cannot be negative." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                      << " " << config_filename << ", TRUE_EMISSION_SERIES" << std::endl;
            std::cerr << std::endl;
            exit(1);
        }
        if (g_eki.true_emissions[i] > 1.0e+20f) {
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Unrealistically large emission at timestep " << (i+1)
                      << ": " << g_eki.true_emissions[i] << " Bq" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Emission rate exceeds physically plausible values." << std::endl;
            std::cerr << "    Check units and magnitude." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Reference:" << Color::RESET << std::endl;
            std::cerr << "    Fukushima accident peak: ~1e+15 - 1e+17 Bq/hour" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                      << " " << config_filename << ", TRUE_EMISSION_SERIES" << std::endl;
            std::cerr << std::endl;
            exit(1);
        }
    }

    // ===== VALIDATION: TIME_INTERVAL =====
    if (g_eki.time_interval <= 0.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid EKI_TIME_INTERVAL: " << g_eki.time_interval << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Time interval must be positive." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    EKI_TIME_INTERVAL > 0.0" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Typical values:" << Color::RESET << std::endl;
        std::cerr << "    - Fine resolution:   5-10 minutes" << std::endl;
        std::cerr << "    - Standard:          15-30 minutes" << std::endl;
        std::cerr << "    - Coarse:            1-3 hours" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << config_filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    std::cout << Color::GREEN << "done" << Color::RESET << std::endl;

    // Print essential EKI settings (condensed)
    std::cout << Color::BOLD << "EKI Configuration" << Color::RESET << std::endl;
    std::cout << "  Receptors          : " << Color::BOLD << g_eki.num_receptors << Color::RESET
              << " (radius: " << g_eki.receptor_capture_radius << "°)" << std::endl;
    std::cout << "  Emission timesteps : " << Color::BOLD << g_eki.true_emissions.size() << Color::RESET
              << " (" << g_eki.time_interval << " " << g_eki.time_unit << ")" << std::endl;
    std::cout << "  Ensemble size      : " << Color::BOLD << g_eki.ensemble_size << Color::RESET << std::endl;

    if (g_eki.memory_doctor_mode) {
        std::cout << "  Memory Doctor      : " << Color::YELLOW << "ON" << Color::RESET << std::endl;
    }
}

// ===========================================================================
// GRID RECEPTOR DEBUG MODE FUNCTIONS
// ===========================================================================

/******************************************************************************
 * @brief Initialize uniform grid of receptors for debugging/validation
 *
 * @details Creates a (2N+1)×(2N+1) square grid of receptors centered at the
 *          emission source location. Used in receptor-debug mode for detailed
 *          spatial analysis of particle dispersion patterns and validation
 *          against analytical solutions.
 *
 *          Grid structure:
 *          - Center: Source location (37°N, 141°E by default)
 *          - Extent: ±N grid points in lat/lon directions
 *          - Spacing: Uniform grid spacing in degrees
 *          - Example: grid_count=5, spacing=0.1° → 11×11=121 receptors
 *
 * @param[in] grid_count_param Grid extent (N receptors in each direction)
 *                             - Typical range: 5-10
 *                             - Total receptors = (2N+1)²
 * @param[in] grid_spacing_param Spacing between receptors (degrees)
 *                               - Typical range: 0.05-0.2°
 *                               - Approximate: 0.1° ≈ 11 km at mid-latitudes
 *
 * @pre CUDA device must be initialized
 * @pre Sufficient GPU memory for receptor arrays
 *
 * @post GPU arrays allocated and initialized:
 *       - d_grid_receptor_lats: Receptor latitudes (degrees N)
 *       - d_grid_receptor_lons: Receptor longitudes (degrees E)
 *       - d_grid_receptor_dose: Dose accumulation (initialized to 0)
 *       - d_grid_receptor_particle_count: Particle counts (initialized to 0)
 * @post Host storage vectors resized:
 *       - grid_receptor_observations
 *       - grid_receptor_particle_counts
 * @post Member variables set:
 *       - grid_count, grid_spacing, grid_receptor_total
 *
 * @algorithm
 *   1. Calculate total receptors = (2*grid_count + 1)²
 *   2. Generate receptor grid centered at source:
 *      for i in [-N, N]:
 *        for j in [-N, N]:
 *          lat = source_lat + i * grid_spacing
 *          lon = source_lon + j * grid_spacing
 *   3. Allocate GPU memory for receptor arrays
 *   4. Copy locations to GPU (hipMemcpy)
 *   5. Initialize dose/count arrays to zero (hipMemset)
 *   6. Resize host storage vectors
 *
 * @note Grid is always square and centered at source location
 * @note Large grids (>20×20) may impact performance due to memory overhead
 * @note Used exclusively in receptor-debug mode (not EKI mode)
 *
 * @memory GPU: 4 * total_receptors * sizeof(float) + 1 * total_receptors * sizeof(int)
 *         Example: 121 receptors = 2.4 KB total
 *
 * @see main_receptor_debug.cu for usage
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::initializeGridReceptors(int grid_count_param, float grid_spacing_param) {
    // Store grid parameters
    grid_count = grid_count_param;
    grid_spacing = grid_spacing_param;
    grid_receptor_total = (2 * grid_count + 1) * (2 * grid_count + 1);

    std::cout << Color::CYAN << "[SYSTEM] " << Color::RESET
              << "Initializing " << Color::BOLD << grid_receptor_total << Color::RESET
              << " grid receptors (" << (2*grid_count+1) << "×" << (2*grid_count+1)
              << ", spacing=" << grid_spacing << "°)" << std::endl;

    // Default source location (Fukushima coordinates)
    float source_lat = 37.0f;
    float source_lon = 141.0f;

    // Prepare host arrays for receptor locations
    std::vector<float> host_lats(grid_receptor_total);
    std::vector<float> host_lons(grid_receptor_total);

    // Generate uniform grid centered at source
    int receptor_idx = 0;
    for (int i = -grid_count; i <= grid_count; i++) {
        for (int j = -grid_count; j <= grid_count; j++) {
            float lat = source_lat + i * grid_spacing;
            float lon = source_lon + j * grid_spacing;

            host_lats[receptor_idx] = lat;
            host_lons[receptor_idx] = lon;
            receptor_idx++;
        }
    }

    // Allocate GPU memory for receptor data
    hipMalloc(&d_grid_receptor_lats, grid_receptor_total * sizeof(float));
    hipMalloc(&d_grid_receptor_lons, grid_receptor_total * sizeof(float));
    hipMalloc(&d_grid_receptor_dose, grid_receptor_total * sizeof(float));
    hipMalloc(&d_grid_receptor_particle_count, grid_receptor_total * sizeof(int));

    // Copy receptor locations to GPU
    hipMemcpy(d_grid_receptor_lats, host_lats.data(), grid_receptor_total * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_receptor_lons, host_lons.data(), grid_receptor_total * sizeof(float), hipMemcpyHostToDevice);

    // Initialize GPU dose and particle count arrays to zero
    hipMemset(d_grid_receptor_dose, 0, grid_receptor_total * sizeof(float));
    hipMemset(d_grid_receptor_particle_count, 0, grid_receptor_total * sizeof(int));

    // Initialize host storage for observations
    grid_receptor_observations.resize(grid_receptor_total);
    grid_receptor_particle_counts.resize(grid_receptor_total);

    std::cout << Color::GREEN << "  " << Color::RESET
              << "Grid receptors initialized" << std::endl;
}

// ===========================================================================
// MODERNIZED CONFIG LOADING FUNCTIONS (Phase 1: 2025-10-17)
// ===========================================================================
// These functions implement the new modular configuration file structure
// described in docs/INPUT_MODERNIZATION_PLAN.md. Provides improved usability
// with self-documenting config files, logical grouping, and backward compatibility.

/******************************************************************************
 * @brief Load simulation parameters from modernized simulation.conf file
 *
 * @details Parses input/simulation.conf to load core simulation settings:
 *          - Temporal: time_end, time_step, vtk_output_frequency
 *          - Particles: total_particles
 *          - Atmosphere: rural_conditions, use_pasquill_gifford
 *          - Meteorology: use_gfs_data
 *          - Terminal: fixed_scroll_output
 *
 *          Part of Phase 1 input file modernization (2025-10-17).
 *
 * @pre input/simulation.conf must exist
 * @post Member variables populated: time_end, dt, freq_output, nop, isRural, isPG, isGFS
 * @post g_sim.fixedScrollOutput set
 * @post Configuration summary printed to console
 *
 * @see docs/INPUT_MODERNIZATION_PLAN.md for config file format
 * @see input/simulation.conf for configuration template
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::loadSimulationConfig() {
    std::cout << Color::CYAN << "[CONFIG] " << Color::RESET
              << "Loading simulation.conf... " << std::flush;

    // Load configuration file
    if (!g_config.loadConfig("input/simulation.conf")) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[FATAL ERROR] "
                  << Color::RESET << "Failed to load input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Possible causes:" << Color::RESET << std::endl;
        std::cerr << "    - File does not exist in the input/ directory" << std::endl;
        std::cerr << "    - Insufficient read permissions" << std::endl;
        std::cerr << "    - File is corrupted or locked by another process" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Solution:" << Color::RESET << std::endl;
        std::cerr << "    - Verify that 'input/simulation.conf' exists" << std::endl;
        std::cerr << "    - Check file permissions: chmod 644 input/simulation.conf" << std::endl;
        std::cerr << "    - Ensure you are running from the project root directory" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ========== TEMPORAL SETTINGS ==========
    // Parse time_end (NO DEFAULT - explicit value required)
    std::string time_end_str = g_config.getString("time_end", "");
    if (time_end_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: time_end" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The time_end parameter is required to define simulation duration." << std::endl;
        std::cerr << "    This sets how long particles are tracked in the atmosphere." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    time_end: <positive number>  # in seconds" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended values:" << Color::RESET << std::endl;
        std::cerr << "    - Short test:      3600  (1 hour)" << std::endl;
        std::cerr << "    - Standard:       21600  (6 hours)" << std::endl;
        std::cerr << "    - Long-range:     86400  (24 hours)" << std::endl;
        std::cerr << "    - Multi-day:     259200  (3 days)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    try {
        time_end = std::stof(time_end_str);
    } catch (const std::exception& e) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Cannot parse time_end value: '" << time_end_str << "'" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Value must be a valid floating-point number." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Valid examples:" << Color::RESET << std::endl;
        std::cerr << "    time_end: 21600" << std::endl;
        std::cerr << "    time_end: 21600.0" << std::endl;
        std::cerr << "    time_end: 2.16e4  # Scientific notation" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // Parse time_step (NO DEFAULT - explicit value required)
    std::string dt_str = g_config.getString("time_step", "");
    if (dt_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: time_step" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The time_step parameter is required for particle advancement." << std::endl;
        std::cerr << "    This controls temporal accuracy vs. computational cost." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    time_step: <positive number>  # in seconds" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended values:" << Color::RESET << std::endl;
        std::cerr << "    - High accuracy:    10-50  seconds (slower)" << std::endl;
        std::cerr << "    - Balanced:        50-100  seconds (recommended)" << std::endl;
        std::cerr << "    - Fast:           100-300  seconds (less accurate)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    try {
        dt = std::stof(dt_str);
    } catch (const std::exception& e) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Cannot parse time_step value: '" << dt_str << "'" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Value must be a valid floating-point number." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Valid examples:" << Color::RESET << std::endl;
        std::cerr << "    time_step: 100" << std::endl;
        std::cerr << "    time_step: 100.0" << std::endl;
        std::cerr << "    time_step: 1.0e2  # Scientific notation" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // Parse vtk_output_frequency (NO DEFAULT - explicit value required)
    std::string freq_str = g_config.getString("vtk_output_frequency", "");
    if (freq_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: vtk_output_frequency" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The vtk_output_frequency parameter controls visualization output." << std::endl;
        std::cerr << "    This determines how often VTK files are written." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    vtk_output_frequency: <positive integer>  # timesteps per output" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended values:" << Color::RESET << std::endl;
        std::cerr << "    - Every timestep:    1  (detailed, large files)" << std::endl;
        std::cerr << "    - Every 10 steps:   10  (balanced)" << std::endl;
        std::cerr << "    - Sparse output:   100  (minimal disk usage)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    try {
        freq_output = std::stoi(freq_str);
    } catch (const std::exception& e) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Cannot parse vtk_output_frequency value: '" << freq_str << "'" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Value must be a valid integer." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Valid examples:" << Color::RESET << std::endl;
        std::cerr << "    vtk_output_frequency: 1" << std::endl;
        std::cerr << "    vtk_output_frequency: 10" << std::endl;
        std::cerr << "    vtk_output_frequency: 100" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: time_end =====
    if (time_end <= 0.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid time_end: " << time_end << " seconds" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Simulation duration must be positive." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    time_end > 0 (in seconds)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended range:" << Color::RESET << std::endl;
        std::cerr << "    - Short test: 3600 (1 hour)" << std::endl;
        std::cerr << "    - Medium: 21600 (6 hours)" << std::endl;
        std::cerr << "    - Long: 86400 (24 hours)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (time_end > 604800.0f) {  // 7 days
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Excessively long simulation time: " << time_end << " seconds ("
                  << (time_end / 86400.0f) << " days)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Simulations longer than 7 days may be impractical due to:" << std::endl;
        std::cerr << "    - Excessive computation time" << std::endl;
        std::cerr << "    - Meteorological data availability" << std::endl;
        std::cerr << "    - Accumulated numerical errors" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    time_end <= 604800 seconds (7 days)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: time_step (dt) =====
    if (dt <= 0.0f) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid time_step: " << dt << " seconds" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Time step must be positive." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    time_step > 0 (in seconds)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended range:" << Color::RESET << std::endl;
        std::cerr << "    - High accuracy: 10-50 seconds" << std::endl;
        std::cerr << "    - Balanced: 50-100 seconds (recommended)" << std::endl;
        std::cerr << "    - Fast: 100-300 seconds (less accurate)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (dt > 3600.0f) {  // 1 hour
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Time step too large: " << dt << " seconds" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Large time steps cause numerical instability and poor accuracy." << std::endl;
        std::cerr << "    Particles may skip over important meteorological features." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    time_step <= 3600 seconds (1 hour)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " 100 seconds for good balance" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (dt >= time_end) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Time step must be smaller than simulation duration" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Current values:" << Color::RESET << std::endl;
        std::cerr << "    time_step = " << dt << " seconds" << std::endl;
        std::cerr << "    time_end  = " << time_end << " seconds" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required:" << Color::RESET << std::endl;
        std::cerr << "    time_step < time_end" << std::endl;
        std::cerr << "    Suggested: time_step <= time_end / 10" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example fix:" << Color::RESET << std::endl;
        std::cerr << "    If time_end = " << time_end << " seconds" << std::endl;
        std::cerr << "    Then time_step = " << (time_end / 100.0f) << " seconds (or smaller)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: vtk_output_frequency =====
    if (freq_output <= 0) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid vtk_output_frequency: " << freq_output << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Output frequency must be positive." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    vtk_output_frequency >= 1" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended values:" << Color::RESET << std::endl;
        std::cerr << "    - Every timestep: 1 (most detailed, large files)" << std::endl;
        std::cerr << "    - Every 10th step: 10 (balanced)" << std::endl;
        std::cerr << "    - Every 100th step: 100 (minimal output)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (freq_output > 1000) {
        std::cerr << std::endl << Color::YELLOW << Color::BOLD << "[WARNING] "
                  << Color::RESET << "Very sparse output frequency: " << freq_output << std::endl;
        std::cerr << "  This may result in insufficient visualization data." << std::endl;
        std::cerr << "  Consider using a smaller value (e.g., 10-100) for better analysis." << std::endl;
        std::cerr << std::endl;
    }

    // ========== PARTICLE SETTINGS ==========
    // Parse total_particles (NO DEFAULT - explicit value required)
    std::string nop_str = g_config.getString("total_particles", "");
    if (nop_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: total_particles" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The total_particles parameter is required for simulation." << std::endl;
        std::cerr << "    This determines statistical quality of results." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    total_particles: <positive integer>  # number of particles" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended values:" << Color::RESET << std::endl;
        std::cerr << "    - Quick test:       1,000  particles" << std::endl;
        std::cerr << "    - Standard:        10,000  particles (good balance)" << std::endl;
        std::cerr << "    - High quality:   100,000  particles" << std::endl;
        std::cerr << "    - Production:   1,000,000  particles (requires GPU)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    try {
        nop = std::stoi(nop_str);
    } catch (const std::exception& e) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Cannot parse total_particles value: '" << nop_str << "'" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Value must be a valid integer." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Valid examples:" << Color::RESET << std::endl;
        std::cerr << "    total_particles: 10000" << std::endl;
        std::cerr << "    total_particles: 100000" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: total_particles =====
    if (nop <= 0) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid total_particles: " << nop << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Particle count must be positive." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    total_particles >= 100 (minimum for meaningful statistics)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended range:" << Color::RESET << std::endl;
        std::cerr << "    - Quick test: 1,000 particles" << std::endl;
        std::cerr << "    - Standard: 10,000 particles (good balance)" << std::endl;
        std::cerr << "    - High quality: 100,000 particles" << std::endl;
        std::cerr << "    - Production: 1,000,000 particles (requires GPU)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (nop < 100) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Too few particles: " << nop << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Fewer than 100 particles produces unreliable statistics." << std::endl;
        std::cerr << "    Results will be dominated by random sampling noise." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    total_particles >= 100" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << " At least 1,000 particles" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    if (nop > 100000000) {  // 100 million
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Excessive particle count: " << nop << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    This many particles will cause:" << std::endl;
        std::cerr << "    - GPU memory exhaustion" << std::endl;
        std::cerr << "    - Extremely long computation times" << std::endl;
        std::cerr << "    - Potential numerical overflow" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required value:" << Color::RESET << std::endl;
        std::cerr << "    total_particles <= 100,000,000" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Typical maximum:" << Color::RESET << " 10,000,000 particles" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // Note: Atmospheric conditions and meteorological data are hardcoded in v1.0
    // isRural=1, isPG=1, isGFS=1 (set in loadLegacyConfig)

    // ========== TERMINAL OUTPUT ==========
    // Parse fixed_scroll_output (NO DEFAULT - explicit value required)
    std::string fixedScroll_str = g_config.getString("fixed_scroll_output", "");
    if (fixedScroll_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: fixed_scroll_output" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The fixed_scroll_output parameter controls terminal output style." << std::endl;
        std::cerr << "    This affects how simulation progress is displayed." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << std::endl;
        std::cerr << "    fixed_scroll_output: <0 or 1>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    0 = Continuous scroll (full history visible)" << std::endl;
        std::cerr << "    1 = Fixed-height (cleaner, stays within terminal)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << std::endl;
        std::cerr << "    fixed_scroll_output: 1  # Cleaner output" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    try {
        g_sim.fixedScrollOutput = std::stoi(fixedScroll_str);
    } catch (const std::exception& e) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Cannot parse fixed_scroll_output value: '" << fixedScroll_str << "'" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Value must be either 0 or 1." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: fixed_scroll_output =====
    if (g_sim.fixedScrollOutput != 0 && g_sim.fixedScrollOutput != 1) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid fixed_scroll_output: " << g_sim.fixedScrollOutput << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    This is a boolean flag - must be 0 or 1." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    0 = Continuous scroll (full history visible)" << std::endl;
        std::cerr << "    1 = Fixed-height (cleaner, stays within terminal)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ========== VTK VISUALIZATION OUTPUT ==========
    // Parse enable_single_mode_vtk (NO DEFAULT - explicit value required)
    std::string vtk_single_str = g_config.getString("enable_single_mode_vtk", "");
    if (vtk_single_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: enable_single_mode_vtk" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The enable_single_mode_vtk parameter controls VTK output for initial simulation." << std::endl;
        std::cerr << "    This affects whether visualization files are generated for the 'truth' run." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << std::endl;
        std::cerr << "    enable_single_mode_vtk: <0 or 1>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    0 = Disable VTK output (faster, no visualization)" << std::endl;
        std::cerr << "    1 = Enable VTK output (useful for visualization)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << std::endl;
        std::cerr << "    enable_single_mode_vtk: 1  # Visualize initial simulation" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    try {
        config_enable_single_mode_vtk = (std::stoi(vtk_single_str) != 0);
    } catch (const std::exception& e) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Cannot parse enable_single_mode_vtk value: '" << vtk_single_str << "'" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Value must be either 0 or 1." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: enable_single_mode_vtk =====
    int vtk_single_int = std::stoi(vtk_single_str);
    if (vtk_single_int != 0 && vtk_single_int != 1) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid enable_single_mode_vtk: " << vtk_single_int << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    This is a boolean flag - must be 0 or 1." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    0 = Disable VTK output (faster, no visualization)" << std::endl;
        std::cerr << "    1 = Enable VTK output (useful for visualization)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // Parse enable_ensemble_mode_vtk (NO DEFAULT - explicit value required)
    std::string vtk_ensemble_str = g_config.getString("enable_ensemble_mode_vtk", "");
    if (vtk_ensemble_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: enable_ensemble_mode_vtk" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The enable_ensemble_mode_vtk parameter controls VTK output for ensemble iterations." << std::endl;
        std::cerr << "    This affects whether visualization files are generated during EKI iterations." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << std::endl;
        std::cerr << "    enable_ensemble_mode_vtk: <0 or 1>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    0 = Disable VTK output (recommended for performance)" << std::endl;
        std::cerr << "    1 = Enable VTK output (large files, slower)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << std::endl;
        std::cerr << "    enable_ensemble_mode_vtk: 0  # Disable for performance" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Note:" << Color::RESET << std::endl;
        std::cerr << "    - Ensemble VTK files can be very large (100s of GB)" << std::endl;
        std::cerr << "    - Only final iteration is saved if enabled" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    try {
        config_enable_ensemble_mode_vtk = (std::stoi(vtk_ensemble_str) != 0);
    } catch (const std::exception& e) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Cannot parse enable_ensemble_mode_vtk value: '" << vtk_ensemble_str << "'" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Value must be either 0 or 1." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET
                  << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: enable_ensemble_mode_vtk =====
    int vtk_ensemble_int = std::stoi(vtk_ensemble_str);
    if (vtk_ensemble_int != 0 && vtk_ensemble_int != 1) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Invalid enable_ensemble_mode_vtk: " << vtk_ensemble_int << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    This is a boolean flag - must be 0 or 1." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    0 = Disable VTK output (recommended for performance)" << std::endl;
        std::cerr << "    1 = Enable VTK output (large files, slower)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/simulation.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    std::cout << Color::GREEN << "done" << Color::RESET << std::endl;

    // ========== PRINT CONFIGURATION SUMMARY ==========
    std::cout << Color::BOLD << "Simulation Configuration" << Color::RESET << std::endl;

    // Temporal settings
    std::cout << "  Time settings      : " << Color::BOLD << time_end << "s" << Color::RESET
              << " (dt=" << dt << "s, "
              << "output_freq=" << freq_output << ")" << std::endl;

    // Particle count
    std::cout << "  Particles          : " << Color::BOLD << nop << Color::RESET << std::endl;

    // Atmospheric conditions
    std::cout << "  Atmosphere         : "
              << (isRural ? "Rural" : "Urban") << ", "
              << (isPG ? "Pasquill-Gifford" : "Briggs-McElroy-Pooler") << std::endl;

    // Meteorological data
    std::cout << "  Meteorology        : " << (isGFS ? "GFS" : "LDAPS") << std::endl;

    // Terminal output
    std::cout << "  Terminal output    : "
              << (g_sim.fixedScrollOutput ? "Fixed-scroll" : "Continuous-scroll") << std::endl;

    // VTK visualization output
    std::cout << "  VTK output         : "
              << "Single=" << (config_enable_single_mode_vtk ? (std::string(Color::GREEN) + "ON") : (std::string(Color::YELLOW) + "OFF")) << Color::RESET
              << ", Ensemble=" << (config_enable_ensemble_mode_vtk ? (std::string(Color::GREEN) + "ON") : (std::string(Color::YELLOW) + "OFF")) << Color::RESET
              << std::endl;
}

/******************************************************************************
 * @brief Load physics model configuration from physics.conf
 *
 * @details Parses input/physics.conf to configure physics model switches:
 *          - dry_deposition_model: Dry deposition (On/Off)
 *          - wet_deposition_model: Wet deposition (On/Off)
 *          - radioactive_decay_model: Decay (On/Off)
 *
 *          Note: turbulence_model is hardcoded to 0 (not implemented in v1.0)
 *
 * @pre input/physics.conf must exist
 * @post Physics switches set: g_drydep, g_wetdep, g_raddecay
 * @post g_turb_switch hardcoded to 0
 * @post Configuration summary printed to console
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::loadPhysicsConfig() {
    std::cout << Color::CYAN << "[SYSTEM] " << Color::RESET
              << "Loading physics configuration... " << std::flush;

    // Load physics.conf using ConfigReader
    ConfigReader physics_config;
    if (!physics_config.loadConfig("input/physics.conf")) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[FATAL ERROR] "
                  << Color::RESET << "Failed to load input/physics.conf" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Possible causes:" << Color::RESET << std::endl;
        std::cerr << "    - File does not exist in the input/ directory" << std::endl;
        std::cerr << "    - Insufficient read permissions" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Solution:" << Color::RESET << std::endl;
        std::cerr << "    - Verify that 'input/physics.conf' exists" << std::endl;
        std::cerr << "    - Check file permissions: chmod 644 input/physics.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // Parse physics model switches (NO DEFAULTS - all must be explicitly provided)
    // Note: turbulence_model is hardcoded to 0 (not implemented in v1.0)

    // Helper lambda to parse On/Off values
    auto parseOnOff = [](const std::string& value, const std::string& param_name) -> int {
        if (value == "On" || value == "on" || value == "ON") {
            return 1;
        } else if (value == "Off" || value == "off" || value == "OFF") {
            return 0;
        } else {
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Invalid value for " << param_name << ": '" << value << "'" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Value must be 'On' or 'Off' (case-insensitive)." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
            std::cerr << "    On  = Enabled" << std::endl;
            std::cerr << "    Off = Disabled" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/physics.conf" << std::endl;
            std::cerr << std::endl;
            exit(1);
        }
        return 0;
    };

    // Parse dry_deposition_model (NO DEFAULT)
    std::string drydep_str = physics_config.getString("dry_deposition_model", "");
    if (drydep_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: dry_deposition_model" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The dry_deposition_model parameter controls gravitational settling." << std::endl;
        std::cerr << "    This is important for particulate matter transport." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    dry_deposition_model: <On or Off>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    Off = Disabled (particles do not settle)" << std::endl;
        std::cerr << "    On  = Enabled (gravitational settling and surface deposition)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << std::endl;
        std::cerr << "    dry_deposition_model: On  # For particulate matter" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/physics.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    g_drydep = parseOnOff(drydep_str, "dry_deposition_model");

    // Parse wet_deposition_model (NO DEFAULT)
    std::string wetdep_str = physics_config.getString("wet_deposition_model", "");
    if (wetdep_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: wet_deposition_model" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The wet_deposition_model parameter controls precipitation removal." << std::endl;
        std::cerr << "    This is critical during rain or snow events." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    wet_deposition_model: <On or Off>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    Off = Disabled (no precipitation removal)" << std::endl;
        std::cerr << "    On  = Enabled (removal by rain and snow)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << std::endl;
        std::cerr << "    wet_deposition_model: On  # If precipitation expected" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/physics.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    g_wetdep = parseOnOff(wetdep_str, "wet_deposition_model");

    // Parse radioactive_decay_model (NO DEFAULT)
    std::string raddecay_str = physics_config.getString("radioactive_decay_model", "");
    if (raddecay_str.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "Missing required parameter: radioactive_decay_model" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    The radioactive_decay_model parameter controls CRAM decay computation." << std::endl;
        std::cerr << "    This is essential for radionuclide transport simulations." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    radioactive_decay_model: <On or Off>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Valid values:" << Color::RESET << std::endl;
        std::cerr << "    Off = Disabled (no radioactive decay)" << std::endl;
        std::cerr << "    On  = Enabled (CRAM decay computation)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Recommended:" << Color::RESET << std::endl;
        std::cerr << "    radioactive_decay_model: On  # Keep ON for radionuclides" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/physics.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }
    g_raddecay = parseOnOff(raddecay_str, "radioactive_decay_model");

    // Note: Validation is handled by parseOnOff() lambda above
    // No additional validation needed - values are guaranteed to be 0 or 1

    std::cout << Color::GREEN << "done" << Color::RESET << std::endl;

    // Print physics model status summary
    std::cout << Color::BOLD << "Physics Models" << Color::RESET << std::endl;
    std::cout << "  Turbulence         : " << (g_turb_switch ? Color::GREEN : Color::YELLOW)
              << (g_turb_switch ? "ON" : "OFF") << Color::RESET << std::endl;
    std::cout << "  Dry Deposition     : " << (g_drydep ? Color::GREEN : Color::YELLOW)
              << (g_drydep ? "ON" : "OFF") << Color::RESET << std::endl;
    std::cout << "  Wet Deposition     : " << (g_wetdep ? Color::GREEN : Color::YELLOW)
              << (g_wetdep ? "ON" : "OFF") << Color::RESET << std::endl;
    std::cout << "  Radioactive Decay  : " << (g_raddecay ? Color::GREEN : Color::YELLOW)
              << (g_raddecay ? "ON" : "OFF") << Color::RESET << std::endl;
}

/******************************************************************************
 * @brief Load source locations from source.conf file
 *
 * @details Parses input/source.conf to load emission source coordinates.
 *          Format: LONGITUDE LATITUDE HEIGHT (space-separated, degrees/meters)
 *
 *          Example:
 *            # Source 1: Fukushima Daiichi
 *            141.0 37.0 20.0
 *
 * @pre input/source.conf must exist
 * @post sources vector populated with Source structs (lon, lat, height)
 * @post At least one source must be defined (validation check)
 *
 * @note Lines starting with # are comments
 * @note Empty lines ignored
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::loadSourceConfig() {
    std::cout << Color::CYAN << "[SYSTEM] " << Color::RESET
              << "Loading source locations... " << std::flush;

    // Construct file path (NO DEFAULT for input_base_path)
    std::string input_base = g_config.getString("input_base_path", "");
    if (input_base.empty()) {
        // Default behavior: use "./input/" if not specified
        // This is acceptable as it's a path convention, not a physical parameter
        input_base = "./input/";
    }
    std::string source_file_path = input_base + "source.conf";

    FILE* sourceFile = fopen(source_file_path.c_str(), "r");

    if (!sourceFile) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[FATAL ERROR] "
                  << Color::RESET << "Failed to open " << source_file_path << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Possible causes:" << Color::RESET << std::endl;
        std::cerr << "    - File does not exist in the input/ directory" << std::endl;
        std::cerr << "    - Insufficient read permissions" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Solution:" << Color::RESET << std::endl;
        std::cerr << "    - Verify that 'input/source.conf' exists" << std::endl;
        std::cerr << "    - Check file permissions: chmod 644 input/source.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    char buffer[256];
    int line_number = 0;

    // Clear existing sources
    sources.clear();

    while (fgets(buffer, sizeof(buffer), sourceFile)) {
        line_number++;

        // Skip comment lines starting with #
        if (buffer[0] == '#') continue;

        // Skip empty lines
        bool is_empty = true;
        for (int i = 0; buffer[i] != '\0'; i++) {
            if (buffer[i] != ' ' && buffer[i] != '\t' &&
                buffer[i] != '\n' && buffer[i] != '\r') {
                is_empty = false;
                break;
            }
        }
        if (is_empty) continue;

        // Stop parsing when encountering a section header (e.g., [GRID_CONFIG])
        // Source locations are defined before any section headers
        if (buffer[0] == '[') {
            break;
        }

        // Parse source location: LON LAT HEIGHT
        Source src;
        int parsed = sscanf(buffer, "%f %f %f", &src.lon, &src.lat, &src.height);

        if (parsed != 3) {
            fclose(sourceFile);
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Invalid format at line " << line_number
                      << " in source.conf" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Invalid line:" << Color::RESET << std::endl;
            std::cerr << "    " << buffer;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
            std::cerr << "    LONGITUDE LATITUDE HEIGHT" << std::endl;
            std::cerr << "    (space-separated, degrees and meters)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
            std::cerr << "    129.48 35.71 100.0" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/source.conf" << std::endl;
            std::cerr << std::endl;
            exit(1);
        }

        // ===== VALIDATION: Longitude =====
        if (src.lon < -180.0f || src.lon > 180.0f) {
            fclose(sourceFile);
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Invalid longitude: " << src.lon << "° at line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Longitude must be in valid geographic range." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Required range:" << Color::RESET << std::endl;
            std::cerr << "    -180.0 <= longitude <= 180.0 (degrees)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Examples:" << Color::RESET << std::endl;
            std::cerr << "    - Tokyo:      139.69°E" << std::endl;
            std::cerr << "    - New York:   -74.01°E (or 285.99°W)" << std::endl;
            std::cerr << "    - Fukushima:  141.00°E" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/source.conf, line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            exit(1);
        }

        // ===== VALIDATION: Latitude =====
        if (src.lat < -90.0f || src.lat > 90.0f) {
            fclose(sourceFile);
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Invalid latitude: " << src.lat << "° at line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Latitude must be in valid geographic range." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Required range:" << Color::RESET << std::endl;
            std::cerr << "    -90.0 <= latitude <= 90.0 (degrees)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Examples:" << Color::RESET << std::endl;
            std::cerr << "    - Equator:    0.00°N" << std::endl;
            std::cerr << "    - Tokyo:      35.69°N" << std::endl;
            std::cerr << "    - Fukushima:  37.00°N" << std::endl;
            std::cerr << "    - South Pole: -90.00°N" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/source.conf, line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            exit(1);
        }

        // ===== VALIDATION: Height =====
        if (src.height < 0.0f) {
            fclose(sourceFile);
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Invalid height: " << src.height << " m at line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Release height cannot be negative (below ground level)." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Required range:" << Color::RESET << std::endl;
            std::cerr << "    height >= 0.0 (meters above ground level)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Typical values:" << Color::RESET << std::endl;
            std::cerr << "    - Ground release:    0-10 m" << std::endl;
            std::cerr << "    - Building release:  20-100 m" << std::endl;
            std::cerr << "    - Stack release:     100-500 m" << std::endl;
            std::cerr << "    - Elevated source:   500-3000 m" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/source.conf, line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            exit(1);
        }
        if (src.height > 20000.0f) {
            fclose(sourceFile);
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Excessive height: " << src.height << " m at line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Release height exceeds practical atmospheric boundary layer." << std::endl;
            std::cerr << "    Heights above 20 km are typically not relevant for dispersion modeling." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Required range:" << Color::RESET << std::endl;
            std::cerr << "    height <= 20000.0 meters (20 km)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Reference:" << Color::RESET << std::endl;
            std::cerr << "    - Troposphere top: ~12 km" << std::endl;
            std::cerr << "    - Stratosphere begins: ~12-15 km" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/source.conf, line "
                      << line_number << std::endl;
            std::cerr << std::endl;
            exit(1);
        }

        sources.push_back(src);
    }

    fclose(sourceFile);

    // Validation: at least one source must be defined
    if (sources.empty()) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "No valid sources found in source.conf" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    At least one emission source must be defined." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Solution:" << Color::RESET << std::endl;
        std::cerr << "    Add at least one source line in the format:" << std::endl;
        std::cerr << "    LONGITUDE LATITUDE HEIGHT" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    # Fukushima Daiichi Nuclear Power Plant" << std::endl;
        std::cerr << "    141.0 37.0 20.0" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " input/source.conf" << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    std::cout << Color::GREEN << "done" << Color::RESET << std::endl;

    // Print loaded sources summary
    std::cout << Color::BOLD << "Source Locations" << Color::RESET << std::endl;
    for (size_t i = 0; i < sources.size(); i++) {
        std::cout << "  Source " << (i+1) << "            : "
                  << sources[i].lon << "°E, "
                  << sources[i].lat << "°N, "
                  << sources[i].height << "m" << std::endl;
    }
}

/******************************************************************************
 * @brief Load nuclide configuration from nuclides.conf
 *
 * @details Parses nuclide properties with backward compatibility for legacy formats.
 *          Tries files in order:
 *          1. input/nuclides.conf (new format)
 *          2. input/nuclides_config_1.txt (legacy single nuclide)
 *          3. input/nuclides_config_60.txt (legacy 60-nuclide chain)
 *
 *          New format (space-separated):
 *            NUCLIDE_NAME DECAY_CONSTANT(s^-1) DEPOSITION_VELOCITY(m/s)
 *
 *          Legacy format (comma-separated):
 *            NUCLIDE_NAME,DECAY_CONSTANT,RATIO
 *
 * @pre At least one nuclide configuration file must exist
 * @post decayConstants vector populated with decay constants (s^-1)
 * @post drydepositionVelocity vector populated with deposition velocities (m/s)
 * @post g_num_nuclides set to number of nuclides loaded
 *
 * @note Decay constants forced to positive values (fabs applied)
 * @note Legacy format uses default deposition velocity = 1.0 m/s
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::loadNuclidesConfig() {
    std::cout << Color::CYAN << "[SYSTEM] " << Color::RESET
              << "Loading nuclide configuration... " << std::flush;

    FILE* nuclideFile = nullptr;
    std::string filename;

    // Try new format first (nuclides.conf)
    filename = "input/nuclides.conf";
    nuclideFile = fopen(filename.c_str(), "r");

    // Fall back to legacy format (nuclides_config_1.txt)
    if (!nuclideFile) {
        filename = "input/nuclides_config_1.txt";
        nuclideFile = fopen(filename.c_str(), "r");
    }

    // Fall back to 60-nuclide chain if available
    if (!nuclideFile) {
        filename = "input/nuclides_config_60.txt";
        nuclideFile = fopen(filename.c_str(), "r");
    }

    if (!nuclideFile) {
        std::cerr << std::endl << Color::RED << "[ERROR] " << Color::RESET
                  << "Cannot open nuclide configuration file" << std::endl;
        std::cerr << "  Tried: input/nuclides.conf" << std::endl;
        std::cerr << "         input/nuclides_config_1.txt" << std::endl;
        std::cerr << "         input/nuclides_config_60.txt" << std::endl;
        exit(1);
    }

    // Clear existing data
    decayConstants.clear();
    drydepositionVelocity.clear();

    char buffer[256];
    int line_number = 0;
    int nuclide_count = 0;

    while (fgets(buffer, sizeof(buffer), nuclideFile)) {
        line_number++;

        // Skip comments and empty lines
        if (buffer[0] == '#' || buffer[0] == '\n' || buffer[0] == '\r') {
            continue;
        }

        // Remove trailing newline
        buffer[strcspn(buffer, "\n\r")] = '\0';

        // Skip empty lines (after trimming)
        if (strlen(buffer) == 0) {
            continue;
        }

        char nuclide_name[64];
        float decay_const, dep_vel;

        // Try new format first (space-separated)
        int parsed = sscanf(buffer, "%s %f %f", nuclide_name, &decay_const, &dep_vel);

        if (parsed == 3) {
            // Successfully parsed new format
            decayConstants.push_back(fabs(decay_const));  // Ensure positive
            drydepositionVelocity.push_back(dep_vel);
            nuclide_count++;
        }
        else {
            // Try legacy comma-separated format
            float legacy_ratio;
            parsed = sscanf(buffer, "%[^,],%f,%f", nuclide_name, &decay_const, &legacy_ratio);

            if (parsed == 3) {
                // Successfully parsed legacy format
                // LEGACY FORMAT WARNING: Using 1.0 m/s as deposition velocity
                // because legacy format doesn't specify it (3rd column is ratio, not dep_vel)
                std::cerr << Color::YELLOW << "[LEGACY FORMAT] " << Color::RESET
                          << "Line " << line_number << ": Using default deposition velocity 1.0 m/s" << std::endl;
                std::cerr << "  Consider migrating to new format: NUCLIDE DECAY_CONST DEP_VEL" << std::endl;
                decayConstants.push_back(fabs(decay_const));  // Ensure positive
                drydepositionVelocity.push_back(1.0f);  // Hardcoded for legacy compatibility
                nuclide_count++;
            }
            else {
                std::cerr << std::endl << Color::YELLOW << "[WARNING] " << Color::RESET
                          << "Failed to parse line " << line_number << " in " << filename << std::endl;
                std::cerr << "  Line: " << buffer << std::endl;
                continue;
            }
        }
    }

    fclose(nuclideFile);

    // Set global nuclide count
    g_num_nuclides = nuclide_count;

    if (nuclide_count == 0) {
        std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                  << Color::RESET << "No valid nuclides loaded from " << filename << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
        std::cerr << "    Nuclide configuration file exists but contains no valid entries." << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Required format:" << Color::RESET << std::endl;
        std::cerr << "    NUCLIDE_NAME DECAY_CONSTANT DEPOSITION_VELOCITY" << std::endl;
        std::cerr << "    (space-separated)" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::GREEN << "Example:" << Color::RESET << std::endl;
        std::cerr << "    Cs137 7.30e-10 0.01" << std::endl;
        std::cerr << "    I131  9.97e-07 0.02" << std::endl;
        std::cerr << std::endl;
        std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << filename << std::endl;
        std::cerr << std::endl;
        exit(1);
    }

    // ===== VALIDATION: Check all decay constants =====
    for (size_t i = 0; i < decayConstants.size(); i++) {
        if (decayConstants[i] < 0.0f) {
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Negative decay constant for nuclide " << (i+1)
                      << ": " << decayConstants[i] << " s⁻¹" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Decay constants must be non-negative." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Physical meaning:" << Color::RESET << std::endl;
            std::cerr << "    Decay constant λ relates to half-life: t₁/₂ = ln(2)/λ" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Typical ranges:" << Color::RESET << std::endl;
            std::cerr << "    - Stable isotopes:  0.0 s⁻¹" << std::endl;
            std::cerr << "    - Long-lived (Cs-137):  7.3e-10 s⁻¹ (t₁/₂ = 30 years)" << std::endl;
            std::cerr << "    - Medium-lived (I-131): 9.97e-07 s⁻¹ (t₁/₂ = 8 days)" << std::endl;
            std::cerr << "    - Short-lived (Xe-133): 1.52e-06 s⁻¹ (t₁/₂ = 5 days)" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << filename << std::endl;
            std::cerr << std::endl;
            exit(1);
        }
        if (decayConstants[i] > 1.0f) {
            std::cerr << std::endl << Color::YELLOW << Color::BOLD << "[WARNING] "
                      << Color::RESET << "Very large decay constant for nuclide " << (i+1)
                      << ": " << decayConstants[i] << " s⁻¹" << std::endl;
            std::cerr << "  This corresponds to a half-life of " << (0.693147f / decayConstants[i])
                      << " seconds." << std::endl;
            std::cerr << "  Such short-lived nuclides may decay before significant transport occurs." << std::endl;
            std::cerr << std::endl;
        }
    }

    // ===== VALIDATION: Check all deposition velocities =====
    for (size_t i = 0; i < drydepositionVelocity.size(); i++) {
        if (drydepositionVelocity[i] < 0.0f) {
            std::cerr << std::endl << Color::RED << Color::BOLD << "[INPUT ERROR] "
                      << Color::RESET << "Negative deposition velocity for nuclide " << (i+1)
                      << ": " << drydepositionVelocity[i] << " m/s" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::YELLOW << "Problem:" << Color::RESET << std::endl;
            std::cerr << "    Deposition velocities must be non-negative." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Physical meaning:" << Color::RESET << std::endl;
            std::cerr << "    Rate at which particles settle to the ground surface." << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::GREEN << "Typical ranges:" << Color::RESET << std::endl;
            std::cerr << "    - Gases:           0.001-0.01 m/s" << std::endl;
            std::cerr << "    - Small particles: 0.001-0.01 m/s" << std::endl;
            std::cerr << "    - Large particles: 0.01-0.1 m/s" << std::endl;
            std::cerr << std::endl;
            std::cerr << "  " << Color::CYAN << "Fix in:" << Color::RESET << " " << filename << std::endl;
            std::cerr << std::endl;
            exit(1);
        }
        if (drydepositionVelocity[i] > 1.0f) {
            std::cerr << std::endl << Color::YELLOW << Color::BOLD << "[WARNING] "
                      << Color::RESET << "Very large deposition velocity for nuclide " << (i+1)
                      << ": " << drydepositionVelocity[i] << " m/s" << std::endl;
            std::cerr << "  Typical deposition velocities are < 0.1 m/s." << std::endl;
            std::cerr << "  Such high values suggest rapid gravitational settling (large particles)." << std::endl;
            std::cerr << std::endl;
        }
    }

    std::cout << Color::GREEN << "done" << Color::RESET << std::endl;

    // Print loaded configuration
    std::cout << Color::BOLD << "Nuclide Configuration" << Color::RESET << std::endl;
    std::cout << "  File               : " << filename << std::endl;
    std::cout << "  Nuclides loaded    : " << Color::BOLD << nuclide_count << Color::RESET << std::endl;

    // Print first nuclide as example
    if (nuclide_count > 0) {
        std::cout << "  Decay constant     : " << decayConstants[0] << " s⁻¹" << std::endl;
        std::cout << "  Deposition velocity: " << drydepositionVelocity[0] << " m/s" << std::endl;
    }
}

/******************************************************************************
 * @brief Load advanced system configuration from advanced.conf
 *
 * @details Validates grid dimensions and coordinate system parameters against
 *          compile-time constants. Provides early warning if config file
 *          dimensions differ from code constants.
 *
 *          Checks:
 *          - gfs_dimX vs Constants::dimX_GFS
 *          - gfs_dimY vs Constants::dimY_GFS
 *          - gfs_dimZ vs Constants::dimZ_GFS
 *
 * @pre input/advanced.conf must exist
 * @post Grid dimensions validated and reported
 *
 * @note Code always uses Constants namespace values (compile-time)
 * @note Dimension mismatch generates warning, not error
 *
 * @author Juryong Park
 * @date 2025
 *****************************************************************************/
void LDM::loadAdvancedConfig() {
    ConfigReader adv_config;

    if (!adv_config.loadConfig("input/advanced.conf")) {
        std::cerr << Color::RED << "[ERROR]" << Color::RESET
                  << " Failed to load input/advanced.conf" << std::endl;
        std::cerr << "This file contains advanced system parameters." << std::endl;
        std::cerr << "If missing, create it using util/generate_config_templates.py" << std::endl;
        exit(1);
    }

    // Load grid dimensions for validation (NO DEFAULTS - must match compile-time constants)
    std::string dimX_str = adv_config.getString("gfs_dimX", "");
    std::string dimY_str = adv_config.getString("gfs_dimY", "");
    std::string dimZ_str = adv_config.getString("gfs_dimZ", "");

    // These parameters are informational/validation only
    // Code always uses Constants namespace values (compile-time)
    // So we allow defaults equal to the compile-time constants
    int cfg_gfs_dimX = dimX_str.empty() ? Constants::dimX_GFS : std::stoi(dimX_str);
    int cfg_gfs_dimY = dimY_str.empty() ? Constants::dimY_GFS : std::stoi(dimY_str);
    int cfg_gfs_dimZ = dimZ_str.empty() ? Constants::dimZ_GFS : std::stoi(dimZ_str);

    // Validate grid dimensions
    bool dimensions_match = (cfg_gfs_dimX == Constants::dimX_GFS) &&
                           (cfg_gfs_dimY == Constants::dimY_GFS) &&
                           (cfg_gfs_dimZ == Constants::dimZ_GFS);

    // Output validation result
    std::cout << Color::BOLD << "Advanced Configuration" << Color::RESET << std::endl;
    std::cout << "  Data paths: " << (isGFS ? "GFS" : "LDAPS") << std::endl;

    if (dimensions_match) {
        std::cout << "  Grid dimensions: " << Color::GREEN << "validated" << Color::RESET << std::endl;
    } else {
        std::cout << "  Grid dimensions: " << Color::YELLOW << "MISMATCH" << Color::RESET << std::endl;
        std::cout << Color::YELLOW << "  Warning: " << Color::RESET
                  << "Config dimensions differ from code constants" << std::endl;
        std::cout << "  Code will use Constants namespace values (compile-time)" << std::endl;
    }
}